#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <host_defines.h>
#include <string.h>
#include <iostream>

#if CUDA_VERSION < 5050
#error CUDA 1.x - 5.0 are not supported any more! Please use CUDA Toolkit 5.5+ instead.
#endif

//#include "log.h"
#include "cudainfo.h"

#define CZ_COPY_BUF_SIZE	(16 * (1 << 20))	/*!< Transfer buffer size. */
#define CZ_COPY_LOOPS_NUM	8			/*!< Number of loops to run transfer test to. */

#define CZ_CALC_BLOCK_LOOPS	16			/*!< Number of loops to run calculation loop. */
#define CZ_CALC_BLOCK_SIZE	256			/*!< Size of instruction block. */
#define CZ_CALC_BLOCK_NUM	16			/*!< Number of instruction blocks in loop. */
#define CZ_CALC_OPS_NUM		2			/*!< Number of operations per one loop. */
#define CZ_CALC_LOOPS_NUM	8			/*!< Number of loops to run performance test to. */

#define CZ_DEF_WARP_SIZE	32			/*!< Default warp size value. */
#define CZ_DEF_THREADS_MAX	512			/*!< Default max threads value value. */

#define CZ_VER_STR_LEN		256			/*!< Version string length. */

#ifndef CZ_LOG_H
#define CZ_LOG_H

#ifdef __cplusplus
extern "C" {
#endif

/*!	\brief Logging level definition.
*/
typedef enum {
	CZLogLevelFatal = -3,		/*!< Fatal error. Causes termination of application. */
	CZLogLevelError = -2,		/*!< Error. */
	CZLogLevelWarning = -1,		/*!< Warning. */
	CZLogLevelHigh = 0,		/*!< Important information. */
	CZLogLevelModerate = 1,		/*!< Moderate information. */
	CZLogLevelLow = 2,		/*!< Not important information. */
} CZLogLevel;

void CZLog(CZLogLevel level, char *fmt, ...){}

#ifdef __cplusplus
}
#endif

#endif//CZ_LOG_H

#define CZ_CUDA_CALL(funcCall, errProc) \
	{ \
		hipError_t errCode; \
		if((errCode = (funcCall)) != hipSuccess) { \
			CZLog(CZLogLevelError, "CUDA Error: %08x %s", errCode, hipGetErrorString(errCode)); \
			errProc; \
		} \
	}

typedef hipError_t (CUDAAPI *cuDeviceGetAttribute_t)(int *pi, hipDeviceAttribute_t attrib, hipDevice_t dev);

typedef hipError_t (CUDAAPI *cuInit_t)(unsigned int Flags);

static cuDeviceGetAttribute_t p_cuDeviceGetAttribute = NULL;
static cuInit_t p_cuInit = NULL;
static char drvVersion[CZ_VER_STR_LEN] = "";
static int drvDllVer = 0;
static char drvDllVerStr[CZ_VER_STR_LEN] = "";
static int rtDllVer = 0;
static char rtDllVerStr[CZ_VER_STR_LEN] = "";

#include <dlfcn.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#define CZ_FILE_STR_LEN		256			/*!< Version file string length. */
#define CZ_VER_FILE_NAME	"/proc/driver/nvidia/version"	/*!< Driver version file name. */
#define CZ_PROC_MAP_NAME	"/proc/self/maps"	/*!< Process memory map file. */
#define CZ_DLL_FNAME		"libcuda.so"		/*!< CUDA dll file name. */
#define CZ_DLL_FNAME_RT		"libcudart.so"		/*!< CUDA RT dll file name. */
#define CZ_LD_SO_CONF		"/etc/ld.so.conf"	/*!< ld.so configuration file. */
#define CZ_LD_SO_DIR		"/etc/ld.so.conf.d/"	/*!< ld.so configuration directory. */
#define CZ_LD_SO_LINE_MAX	100			/*!< ld.so configuration line length. */

static char *CZGetSoVersion(
	char *name,			/*!<[in] Name of so file. E.g. "libcuda.so". */
	char *version			/*!<[out] Library version buffer. */
) {
	FILE *fp = NULL;
	char str[CZ_FILE_STR_LEN];
	int found = 0;

	fp = fopen(CZ_PROC_MAP_NAME, "r");
	if(fp == NULL) {
		return NULL;
	}

	while(fgets(str, CZ_FILE_STR_LEN - 1, fp) != NULL) {
		if(strstr(str, name) != NULL) {
			char *p = NULL;
			char fname[CZ_FILE_STR_LEN];

			p = str + strlen(str) - 1;
			while((p >= str) && ((*p == ' ') || (*p == '\n') || (*p == '\r') || (*p == '\t') || (*p == 0))) {
				*p = 0;
				p--;
			}

			while((p >= str) && ((*p != ' ') && (*p != '\n') && (*p != '\r') && (*p != '\t') && (*p != 0))) {
				p--;
			}

			strncpy(fname, p, CZ_FILE_STR_LEN - 1);
			p = basename(fname);
			if(p == NULL)
				continue;

			if(strstr(p, name) != p) {
				continue;
			}

			p = p + strlen(name);
			if(*p != '.')
				continue;

			strncpy(version, p + 1, CZ_VER_STR_LEN - 1);

			found++;
			break;
		}
	}

	fclose(fp);

	if(found) {
		//CZLog(CZLogLevelLow, "Version of %s is %s.", name, version);
		return version;
	} else {
		return NULL;
	}
}

static bool CZCudaIsInit(void) {

	void *hDll = NULL;

	if((p_cuInit == NULL) || (p_cuDeviceGetAttribute == NULL)) {

		if(hDll == NULL) {
			hDll = dlopen(CZ_DLL_FNAME, RTLD_LAZY);
		}

		if(hDll == NULL) {
			hDll = dlopen("/usr/lib/" CZ_DLL_FNAME, RTLD_LAZY);
		}

		if(hDll == NULL) {
			hDll = dlopen("/usr/lib/nvidia-current/" CZ_DLL_FNAME, RTLD_LAZY);
		}

		if(hDll == NULL) {
			hDll = dlopen("/usr/lib32/" CZ_DLL_FNAME, RTLD_LAZY);
		}

		if(hDll == NULL) {
			hDll = dlopen("/usr/lib32/nvidia-current/" CZ_DLL_FNAME, RTLD_LAZY);
		}

		if(hDll == NULL) {
			hDll = dlopen("/usr/lib64/" CZ_DLL_FNAME, RTLD_LAZY);
		}

		if(hDll == NULL) {
			hDll = dlopen("/usr/lib64/nvidia-current/" CZ_DLL_FNAME, RTLD_LAZY);
		}

		if(hDll == NULL) {
			hDll = dlopen("/usr/lib128/" CZ_DLL_FNAME, RTLD_LAZY);
		}

		if(hDll == NULL) {
			hDll = dlopen("/usr/lib128/nvidia-current/" CZ_DLL_FNAME, RTLD_LAZY);
		}

		// Try to check ld.so.conf & Co
		if(hDll == NULL) {
			char buf[CZ_LD_SO_LINE_MAX + sizeof(CZ_DLL_FNAME) + 1];
			char *p;
			FILE *f;

			f = popen("cat " CZ_LD_SO_CONF " " CZ_LD_SO_DIR "/*", "r");
			if(f != NULL) {
				while(fgets(buf, CZ_LD_SO_LINE_MAX, f) != NULL) {
					if((p = strchr(buf, '\n')) != NULL) *p = 0;
					if((p = strchr(buf, '#')) != NULL) *p = 0;
					if(strlen(buf) > 0) {
						strcat(buf, "/" CZ_DLL_FNAME);
						if(hDll == NULL) {
							hDll = dlopen(buf, RTLD_LAZY);
						}
					}
				}
				pclose(f);
			}
		}

		if(hDll == NULL) {
			//CZLog(CZLogLevelError, "Can't load CUDA driver.");
			return false;
		}

		p_cuDeviceGetAttribute = (cuDeviceGetAttribute_t)dlsym(hDll, "hipDeviceGetAttribute");
		if(p_cuDeviceGetAttribute == NULL) {
			return false;
		}

		p_cuInit = (cuInit_t)dlsym(hDll, "hipInit");
		if(p_cuInit == NULL) {
			return false;
		}

		CZGetSoVersion(CZ_DLL_FNAME, drvDllVerStr);
		CZGetSoVersion(CZ_DLL_FNAME_RT, rtDllVerStr);

		if(access(CZ_VER_FILE_NAME, R_OK) == 0) {
			FILE *fp = NULL;
			char str[CZ_FILE_STR_LEN];
			fp = fopen(CZ_VER_FILE_NAME, "r");
			if(fp != NULL) {
				while(fgets(str, CZ_FILE_STR_LEN - 1, fp) != NULL) {
					char *p = NULL;
					char *kernel_module = "Kernel Module";
					if((p = strstr(str, kernel_module)) != NULL) {
						p += strlen(kernel_module);
						while(*p == ' ')
							p++;
						strncpy(drvVersion, p, CZ_VER_STR_LEN - 1);
						p = drvVersion;
						while((*p != ' ') && (*p != '\n') && (*p != '\r') && (*p != '\t') && (*p != 0)) {
							p++;
						}
						*p = 0;
						break;
					}
				}
				fclose(fp);
			}
		}

	}
	return true;
}

bool CZCudaCheck(void) {

	if(!CZCudaIsInit())
		return false;

	if(p_cuInit(0) == hipErrorNotInitialized) {
		return false;
	}

	CZ_CUDA_CALL(hipDriverGetVersion(&drvDllVer),
		drvDllVer = 0);

	//CZLog(CZLogLevelLow, "Driver version %d.", drvDllVer);

	CZ_CUDA_CALL(hipRuntimeGetVersion(&rtDllVer),
		rtDllVer = 0);

	//CZLog(CZLogLevelLow, "Runtime version %d.", rtDllVer);

	return true;
}

int CZCudaDeviceFound(void) {

	int count;

	CZ_CUDA_CALL(hipGetDeviceCount(&count),
		return 0);

	return count;
}

#define ConvertSMVer2Cores(major, minor) \
	(((major) == 1)? ( /* Tesla */ \
		((minor) == 0)? 8: /* G80*/ \
		((minor) == 1)? 8: /* G8x */ \
		((minor) == 2)? 8: /* G9x */ \
		((minor) == 3)? 8: /* GT200 */ \
		0): \
	((major) == 2)? ( /* Fermi */ \
		((minor) == 0)? 32: /* GF100 */ \
		((minor) == 1)? 48: /* GF10x */ \
		0): \
	((major) == 3)? ( /* Kepler */ \
		((minor) == 0)? 192: /* GK10x */ \
		((minor) == 2)? 192: /* GK10x */ \
		((minor) == 5)? 192: /* GK11x */ \
		((minor) == 7)? 192: /* GK21x */ \
		0): \
	((major) == 5)? ( /* Maxwell */ \
		((minor) == 0)? 128: /* GM10X */ \
		((minor) == 2)? 128: /* GM20X */ \
		0): \
	0)

#define COMPILE_ASSERT(cond)	{typedef char compile_assert_error[(cond)? 1: -1];}

int CZCudaReadDeviceInfo(
	struct CZDeviceInfo *info,	/*!<[in,out] CUDA-device information. */
	int num				/*!<[in] Number (index) of CUDA-device. */
) {
	hipDeviceProp_t prop;
//	int ecc;

	if(info == NULL)
		return -1;

	if(!CZCudaIsInit())
		return -1;

	if(num >= CZCudaDeviceFound())
		return -1;

	CZ_CUDA_CALL(hipGetDeviceProperties(&prop, num),
		return -1);

	info->num = num;
	strcpy(info->deviceName, prop.name);
	info->major = prop.major;
	info->minor = prop.minor;
	info->drvVersion = drvVersion;
	info->drvDllVer = drvDllVer;
	info->drvDllVerStr = drvDllVerStr;
	info->rtDllVer = rtDllVer;
	info->rtDllVerStr = rtDllVerStr;
	info->tccDriver = prop.tccDriver;

	info->core.regsPerBlock = prop.regsPerBlock;
	info->core.SIMDWidth = prop.warpSize;
	info->core.maxThreadsPerBlock = prop.maxThreadsPerBlock;
	info->core.maxThreadsDim[0] = prop.maxThreadsDim[0];
	info->core.maxThreadsDim[1] = prop.maxThreadsDim[1];
	info->core.maxThreadsDim[2] = prop.maxThreadsDim[2];
	info->core.maxGridSize[0] = prop.maxGridSize[0];
	info->core.maxGridSize[1] = prop.maxGridSize[1];
	info->core.maxGridSize[2] = prop.maxGridSize[2];
	info->core.clockRate = prop.clockRate;
	info->core.muliProcCount = prop.multiProcessorCount;
	info->core.watchdogEnabled = prop.kernelExecTimeoutEnabled;
	info->core.integratedGpu = prop.integrated;
	info->core.concurrentKernels = prop.concurrentKernels;
	info->core.computeMode =
		(prop.computeMode == hipComputeModeDefault)? CZComputeModeDefault:
		(prop.computeMode == hipComputeModeExclusive)? CZComputeModeExclusive:
		(prop.computeMode == hipComputeModeProhibited)? CZComputeModeProhibited:
		CZComputeModeUnknown;
	info->core.pciBusID = prop.pciBusID;
	info->core.pciDeviceID = prop.pciDeviceID;
	info->core.pciDomainID = prop.pciDomainID;
	info->core.maxThreadsPerMultiProcessor = prop.maxThreadsPerMultiProcessor;
	info->core.cudaCores = ConvertSMVer2Cores(prop.major, prop.minor) * prop.multiProcessorCount;
	info->core.streamPrioritiesSupported = prop.streamPrioritiesSupported;

	info->mem.totalGlobal = prop.totalGlobalMem;
	info->mem.sharedPerBlock = prop.sharedMemPerBlock;
	info->mem.maxPitch = prop.memPitch;
	info->mem.totalConst = prop.totalConstMem;
	info->mem.textureAlignment = prop.textureAlignment;
	info->mem.texture1D[0] = prop.maxTexture1D;
	info->mem.texture2D[0] = prop.maxTexture2D[0];
	info->mem.texture2D[1] = prop.maxTexture2D[1];
	info->mem.texture3D[0] = prop.maxTexture3D[0];
	info->mem.texture3D[1] = prop.maxTexture3D[1];
	info->mem.texture3D[2] = prop.maxTexture3D[2];
	info->mem.gpuOverlap = prop.deviceOverlap;
	info->mem.mapHostMemory = prop.canMapHostMemory;
        info->mem.errorCorrection = prop.ECCEnabled;
	info->mem.asyncEngineCount = prop.asyncEngineCount;
	info->mem.unifiedAddressing = prop.unifiedAddressing;
	info->mem.memoryClockRate = prop.memoryClockRate;
	info->mem.memoryBusWidth = prop.memoryBusWidth;
	info->mem.l2CacheSize = prop.l2CacheSize;

	return 0;
}

struct CZDeviceInfoBandLocalData {
	void		*memHostPage;	/*!< Pageable host memory. */
	void		*memHostPin;	/*!< Pinned host memory. */
	void		*memDevice1;	/*!< Device memory buffer 1. */
	void		*memDevice2;	/*!< Device memory buffer 2. */
};

int CZCudaCalcDeviceSelect(
	struct CZDeviceInfo *info	/*!<[in,out] CUDA-device information. */
) {

	//CZLog(CZLogLevelLow, "Selecting %s.", info->deviceName);

	CZ_CUDA_CALL(hipSetDevice(info->num),
		return -1);

	return 0;
}

static int CZCudaCalcDeviceBandwidthAlloc(
	struct CZDeviceInfo *info	/*!<[in,out] CUDA-device information. */
) {
	CZDeviceInfoBandLocalData *lData;

	if(info == NULL)
		return -1;

	if(info->band.localData == NULL) {

		//CZLog(CZLogLevelLow, "Alloc local buffers for %s.", info->deviceName);

		lData = (CZDeviceInfoBandLocalData*)malloc(sizeof(*lData));
		if(lData == NULL) {
			return -1;
		}

		//CZLog(CZLogLevelLow, "Alloc host pageable for %s.", info->deviceName);

		lData->memHostPage = (void*)malloc(CZ_COPY_BUF_SIZE);
		if(lData->memHostPage == NULL) {
			free(lData);
			return -1;
		}

		//CZLog(CZLogLevelLow, "Host pageable is at 0x%08X.", lData->memHostPage);

		//CZLog(CZLogLevelLow, "Alloc host pinned for %s.", info->deviceName);

		CZ_CUDA_CALL(hipHostMalloc((void**)&lData->memHostPin, CZ_COPY_BUF_SIZE),
			free(lData->memHostPage);
			free(lData);
			return -1);

		//CZLog(CZLogLevelLow, "Host pinned is at 0x%08X.", lData->memHostPin);

		//CZLog(CZLogLevelLow, "Alloc device buffer 1 for %s.", info->deviceName);

		CZ_CUDA_CALL(hipMalloc((void**)&lData->memDevice1, CZ_COPY_BUF_SIZE),
			hipHostFree(lData->memHostPin);
			free(lData->memHostPage);
			free(lData);
			return -1);

		//CZLog(CZLogLevelLow, "Device buffer 1 is at 0x%08X.", lData->memDevice1);

		//CZLog(CZLogLevelLow, "Alloc device buffer 2 for %s.", info->deviceName);

		CZ_CUDA_CALL(hipMalloc((void**)&lData->memDevice2, CZ_COPY_BUF_SIZE),
			hipFree(lData->memDevice1);
			hipHostFree(lData->memHostPin);
			free(lData->memHostPage);
			free(lData);
			return -1);

		CZLog(CZLogLevelLow, "Device buffer 2 is at 0x%08X.", lData->memDevice2);

		info->band.localData = (void*)lData;
	}

	return 0;
}

static int CZCudaCalcDeviceBandwidthFree(
	struct CZDeviceInfo *info	/*!<[in,out] CUDA-device information. */
) {
	CZDeviceInfoBandLocalData *lData;

	if(info == NULL)
		return -1;

	lData = (CZDeviceInfoBandLocalData*)info->band.localData;
	if(lData != NULL) {

		//CZLog(CZLogLevelLow, "Free host pageable for %s.", info->deviceName);

		if(lData->memHostPage != NULL)
			free(lData->memHostPage);

		//CZLog(CZLogLevelLow, "Free host pinned for %s.", info->deviceName);

		if(lData->memHostPin != NULL)
			hipHostFree(lData->memHostPin);

		//CZLog(CZLogLevelLow, "Free device buffer 1 for %s.", info->deviceName);

		if(lData->memDevice1 != NULL)
			hipFree(lData->memDevice1);

		//CZLog(CZLogLevelLow, "Free device buffer 2 for %s.", info->deviceName);

		if(lData->memDevice2 != NULL)
			hipFree(lData->memDevice2);

		//CZLog(CZLogLevelLow, "Free local buffers for %s.", info->deviceName);

		free(lData);
	}
	info->band.localData = NULL;

	return 0;
}

static int CZCudaCalcDeviceBandwidthReset(
	struct CZDeviceInfo *info	/*!<[in,out] CUDA-device information. */
) {

	if(info == NULL)
		return -1;

	info->band.copyHDPage = 0;
	info->band.copyHDPin = 0;
	info->band.copyDHPage = 0;
	info->band.copyDHPin = 0;
	info->band.copyDD = 0;

	return 0;
}

#define CZ_COPY_MODE_H2D	0	/*!< Host to device data copy mode. */
#define CZ_COPY_MODE_D2H	1	/*!< Device to host data copy mode. */
#define CZ_COPY_MODE_D2D	2	/*!< Device to device data copy mode. */

static float CZCudaCalcDeviceBandwidthTestCommon (
	struct CZDeviceInfo *info,	/*!<[in,out] CUDA-device information. */
	int mode,			/*!<[in] Run bandwidth test in one of modes. */
	int pinned			/*!<[in] Use pinned \a (=1) memory buffer instead of pagable \a (=0). */
) {
	CZDeviceInfoBandLocalData *lData;
	float timeMs = 0.0;
	float bandwidthKiBs = 0.0;
	hipEvent_t start;
	hipEvent_t stop;
	void *memHost;
	void *memDevice1;
	void *memDevice2;
	int i;

	if(info == NULL)
		return 0;

	CZ_CUDA_CALL(hipEventCreate(&start),
		return 0);

	CZ_CUDA_CALL(hipEventCreate(&stop),
		hipEventDestroy(start);
		return 0);

	lData = (CZDeviceInfoBandLocalData*)info->band.localData;

	memHost = pinned? lData->memHostPin: lData->memHostPage;
	memDevice1 = lData->memDevice1;
	memDevice2 = lData->memDevice2;

	CZLog(CZLogLevelLow, "Starting %s test (%s) on %s.",
		(mode == CZ_COPY_MODE_H2D)? "host to device":
		(mode == CZ_COPY_MODE_D2H)? "device to host":
		(mode == CZ_COPY_MODE_D2D)? "device to device": "unknown",
		pinned? "pinned": "pageable",
		info->deviceName);

	for(i = 0; i < CZ_COPY_LOOPS_NUM; i++) {

		float loopMs = 0.0;

		CZ_CUDA_CALL(hipEventRecord(start, 0),
			hipEventDestroy(start);
			hipEventDestroy(stop);
			return 0);

		switch(mode) {
		case CZ_COPY_MODE_H2D:
			CZ_CUDA_CALL(hipMemcpy(memDevice1, memHost, CZ_COPY_BUF_SIZE, hipMemcpyHostToDevice),
				hipEventDestroy(start);
				hipEventDestroy(stop);
				return 0);
			break;

		case CZ_COPY_MODE_D2H:
			CZ_CUDA_CALL(hipMemcpy(memHost, memDevice2, CZ_COPY_BUF_SIZE, hipMemcpyDeviceToHost),
				hipEventDestroy(start);
				hipEventDestroy(stop);
				return 0);
			break;

		case CZ_COPY_MODE_D2D:
			CZ_CUDA_CALL(hipMemcpy(memDevice2, memDevice1, CZ_COPY_BUF_SIZE, hipMemcpyDeviceToDevice),
				hipEventDestroy(start);
				hipEventDestroy(stop);
				return 0);
			break;

		default: // WTF!
			hipEventDestroy(start);
			hipEventDestroy(stop);
			return 0;
		}

		CZ_CUDA_CALL(hipEventRecord(stop, 0),
			hipEventDestroy(start);
			hipEventDestroy(stop);
			return 0);

		CZ_CUDA_CALL(hipEventSynchronize(stop),
			hipEventDestroy(start);
			hipEventDestroy(stop);
			return 0);

		CZ_CUDA_CALL(hipEventElapsedTime(&loopMs, start, stop),
			hipEventDestroy(start);
			hipEventDestroy(stop);
			return 0);

		timeMs += loopMs;
	}

	//CZLog(CZLogLevelLow, "Test complete in %f ms.", timeMs);

	bandwidthKiBs = (
		1000 *
		(float)CZ_COPY_BUF_SIZE *
		(float)CZ_COPY_LOOPS_NUM
	) / (
		timeMs *
		(float)(1 << 10)
	);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return bandwidthKiBs;
}

static int CZCudaCalcDeviceBandwidthTest(
	struct CZDeviceInfo *info	/*!<[in,out] CUDA-device information. */
) {

	info->band.copyHDPage = CZCudaCalcDeviceBandwidthTestCommon(info, CZ_COPY_MODE_H2D, 0);
	info->band.copyHDPin = CZCudaCalcDeviceBandwidthTestCommon(info, CZ_COPY_MODE_H2D, 1);
	info->band.copyDHPage = CZCudaCalcDeviceBandwidthTestCommon(info, CZ_COPY_MODE_D2H, 0);
	info->band.copyDHPin = CZCudaCalcDeviceBandwidthTestCommon(info, CZ_COPY_MODE_D2H, 1);
	info->band.copyDD = CZCudaCalcDeviceBandwidthTestCommon(info, CZ_COPY_MODE_D2D, 0);

	return 0;
}

int CZCudaPrepareDevice(
	struct CZDeviceInfo *info	/*!<[in,out] CUDA-device information. */
) {

	if(info == NULL)
		return -1;

	if(!CZCudaIsInit())
		return -1;

	if(CZCudaCalcDeviceBandwidthAlloc(info) != 0)
		return -1;

	return 0;
}

int CZCudaCalcDeviceBandwidth(
	struct CZDeviceInfo *info	/*!<[in,out] CUDA-device information. */
) {

	if(info == NULL)
		return -1;

	if(CZCudaCalcDeviceBandwidthReset(info) != 0)
		return -1;

	if(!CZCudaIsInit())
		return -1;

	if(CZCudaCalcDeviceBandwidthAlloc(info) != 0)
		return -1;

	if(CZCudaCalcDeviceBandwidthTest(info) != 0)
		return -1;

	return 0;
}

int CZCudaCleanDevice(
	struct CZDeviceInfo *info	/*!<[in,out] CUDA-device information. */
) {

	if(info == NULL)
		return -1;

	if(CZCudaCalcDeviceBandwidthFree(info) != 0)
		return -1;

	return 0;
}

static int CZCudaCalcDevicePerformanceReset(
	struct CZDeviceInfo *info	/*!<[in,out] CUDA-device information. */
) {

	if(info == NULL)
		return -1;

	info->perf.calcFloat = 0;
	info->perf.calcDouble = 0;
	info->perf.calcInteger32 = 0;
	info->perf.calcInteger24 = 0;
	info->perf.calcInteger64 = 0;

	return 0;
}

#define CZ_CALC_FMAD_16(a, b) \
	a = a * a + a; b = b * b + b; a = a * a + a; b = b * b + b; \
	a = a * a + a; b = b * b + b; a = a * a + a; b = b * b + b; \
	a = a * a + a; b = b * b + b; a = a * a + a; b = b * b + b; \
	a = a * a + a; b = b * b + b; a = a * a + a; b = b * b + b; \

#define CZ_CALC_FMAD_256(a, b) \
	CZ_CALC_FMAD_16(a, b) CZ_CALC_FMAD_16(a, b) \
	CZ_CALC_FMAD_16(a, b) CZ_CALC_FMAD_16(a, b) \
	CZ_CALC_FMAD_16(a, b) CZ_CALC_FMAD_16(a, b) \
	CZ_CALC_FMAD_16(a, b) CZ_CALC_FMAD_16(a, b) \
	CZ_CALC_FMAD_16(a, b) CZ_CALC_FMAD_16(a, b) \
	CZ_CALC_FMAD_16(a, b) CZ_CALC_FMAD_16(a, b) \
	CZ_CALC_FMAD_16(a, b) CZ_CALC_FMAD_16(a, b) \
	CZ_CALC_FMAD_16(a, b) CZ_CALC_FMAD_16(a, b) \

#define CZ_CALC_DFMAD_16(a, b) \
	a = a * a + a; b = b * b + b; a = a * a + a; b = b * b + b; \
	a = a * a + a; b = b * b + b; a = a * a + a; b = b * b + b; \
	a = a * a + a; b = b * b + b; a = a * a + a; b = b * b + b; \
	a = a * a + a; b = b * b + b; a = a * a + a; b = b * b + b; \

#define CZ_CALC_DFMAD_256(a, b) \
	CZ_CALC_DFMAD_16(a, b) CZ_CALC_DFMAD_16(a, b) \
	CZ_CALC_DFMAD_16(a, b) CZ_CALC_DFMAD_16(a, b) \
	CZ_CALC_DFMAD_16(a, b) CZ_CALC_DFMAD_16(a, b) \
	CZ_CALC_DFMAD_16(a, b) CZ_CALC_DFMAD_16(a, b) \
	CZ_CALC_DFMAD_16(a, b) CZ_CALC_DFMAD_16(a, b) \
	CZ_CALC_DFMAD_16(a, b) CZ_CALC_DFMAD_16(a, b) \
	CZ_CALC_DFMAD_16(a, b) CZ_CALC_DFMAD_16(a, b) \
	CZ_CALC_DFMAD_16(a, b) CZ_CALC_DFMAD_16(a, b) \

#define CZ_CALC_IMAD32_16(a, b) \
	a = a * a + a; b = b * b + b; a = a * a + a; b = b * b + b; \
	a = a * a + a; b = b * b + b; a = a * a + a; b = b * b + b; \
	a = a * a + a; b = b * b + b; a = a * a + a; b = b * b + b; \
	a = a * a + a; b = b * b + b; a = a * a + a; b = b * b + b; \

#define CZ_CALC_IMAD32_256(a, b) \
	CZ_CALC_IMAD32_16(a, b) CZ_CALC_IMAD32_16(a, b) \
	CZ_CALC_IMAD32_16(a, b) CZ_CALC_IMAD32_16(a, b) \
	CZ_CALC_IMAD32_16(a, b) CZ_CALC_IMAD32_16(a, b) \
	CZ_CALC_IMAD32_16(a, b) CZ_CALC_IMAD32_16(a, b) \
	CZ_CALC_IMAD32_16(a, b) CZ_CALC_IMAD32_16(a, b) \
	CZ_CALC_IMAD32_16(a, b) CZ_CALC_IMAD32_16(a, b) \
	CZ_CALC_IMAD32_16(a, b) CZ_CALC_IMAD32_16(a, b) \
	CZ_CALC_IMAD32_16(a, b) CZ_CALC_IMAD32_16(a, b) \

#define CZ_CALC_IMAD64_16(a, b) \
	a = a * a + a; b = b * b + b; a = a * a + a; b = b * b + b; \
	a = a * a + a; b = b * b + b; a = a * a + a; b = b * b + b; \
	a = a * a + a; b = b * b + b; a = a * a + a; b = b * b + b; \
	a = a * a + a; b = b * b + b; a = a * a + a; b = b * b + b; \

#define CZ_CALC_IMAD64_256(a, b) \
	CZ_CALC_IMAD64_16(a, b) CZ_CALC_IMAD64_16(a, b) \
	CZ_CALC_IMAD64_16(a, b) CZ_CALC_IMAD64_16(a, b) \
	CZ_CALC_IMAD64_16(a, b) CZ_CALC_IMAD64_16(a, b) \
	CZ_CALC_IMAD64_16(a, b) CZ_CALC_IMAD64_16(a, b) \
	CZ_CALC_IMAD64_16(a, b) CZ_CALC_IMAD64_16(a, b) \
	CZ_CALC_IMAD64_16(a, b) CZ_CALC_IMAD64_16(a, b) \
	CZ_CALC_IMAD64_16(a, b) CZ_CALC_IMAD64_16(a, b) \
	CZ_CALC_IMAD64_16(a, b) CZ_CALC_IMAD64_16(a, b) \

#define CZ_CALC_IMAD24_16(a, b) \
	a = __mul24(a, a) + a; b = __mul24(b, b) + b; \
	a = __mul24(a, a) + a; b = __mul24(b, b) + b; \
	a = __mul24(a, a) + a; b = __mul24(b, b) + b; \
	a = __mul24(a, a) + a; b = __mul24(b, b) + b; \
	a = __mul24(a, a) + a; b = __mul24(b, b) + b; \
	a = __mul24(a, a) + a; b = __mul24(b, b) + b; \
	a = __mul24(a, a) + a; b = __mul24(b, b) + b; \
	a = __mul24(a, a) + a; b = __mul24(b, b) + b; \

#define CZ_CALC_IMAD24_256(a, b) \
	CZ_CALC_IMAD24_16(a, b) CZ_CALC_IMAD24_16(a, b)\
	CZ_CALC_IMAD24_16(a, b) CZ_CALC_IMAD24_16(a, b)\
	CZ_CALC_IMAD24_16(a, b) CZ_CALC_IMAD24_16(a, b)\
	CZ_CALC_IMAD24_16(a, b) CZ_CALC_IMAD24_16(a, b)\
	CZ_CALC_IMAD24_16(a, b) CZ_CALC_IMAD24_16(a, b)\
	CZ_CALC_IMAD24_16(a, b) CZ_CALC_IMAD24_16(a, b)\
	CZ_CALC_IMAD24_16(a, b) CZ_CALC_IMAD24_16(a, b)\
	CZ_CALC_IMAD24_16(a, b) CZ_CALC_IMAD24_16(a, b)\

#define CZ_CALC_MODE_FLOAT	0	/*!< Single-precision float point test mode. */
#define CZ_CALC_MODE_DOUBLE	1	/*!< Double-precision float point test mode. */
#define CZ_CALC_MODE_INTEGER32	2	/*!< 32-bit integer test mode. */
#define CZ_CALC_MODE_INTEGER24	3	/*!< 24-bit integer test mode. */
#define CZ_CALC_MODE_INTEGER64	4	/*!< 64-bit integer test mode. */

__global__ void CZCudaCalcKernelFloat(
	void *buf			/*!<[in] Data buffer. */
) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	float *arr = (float*)buf;
	float val1 = index;
	float val2 = arr[index];
	int i;

	for(i = 0; i < CZ_CALC_BLOCK_LOOPS; i++) {
		CZ_CALC_FMAD_256(val1, val2);
		CZ_CALC_FMAD_256(val1, val2);
		CZ_CALC_FMAD_256(val1, val2);
		CZ_CALC_FMAD_256(val1, val2);
		CZ_CALC_FMAD_256(val1, val2);
		CZ_CALC_FMAD_256(val1, val2);
		CZ_CALC_FMAD_256(val1, val2);
		CZ_CALC_FMAD_256(val1, val2);
		CZ_CALC_FMAD_256(val1, val2);
		CZ_CALC_FMAD_256(val1, val2);
		CZ_CALC_FMAD_256(val1, val2);
		CZ_CALC_FMAD_256(val1, val2);
		CZ_CALC_FMAD_256(val1, val2);
		CZ_CALC_FMAD_256(val1, val2);
		CZ_CALC_FMAD_256(val1, val2);
		CZ_CALC_FMAD_256(val1, val2);
	}

	arr[index] = val1 + val2;
}

__global__ void CZCudaCalcKernelDouble(
	void *buf			/*!<[in] Data buffer. */
) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	double *arr = (double*)buf;
	double val1 = index;
	double val2 = arr[index];
	int i;

	for(i = 0; i < CZ_CALC_BLOCK_LOOPS; i++) {
		CZ_CALC_DFMAD_256(val1, val2);
		CZ_CALC_DFMAD_256(val1, val2);
		CZ_CALC_DFMAD_256(val1, val2);
		CZ_CALC_DFMAD_256(val1, val2);
		CZ_CALC_DFMAD_256(val1, val2);
		CZ_CALC_DFMAD_256(val1, val2);
		CZ_CALC_DFMAD_256(val1, val2);
		CZ_CALC_DFMAD_256(val1, val2);
		CZ_CALC_DFMAD_256(val1, val2);
		CZ_CALC_DFMAD_256(val1, val2);
		CZ_CALC_DFMAD_256(val1, val2);
		CZ_CALC_DFMAD_256(val1, val2);
		CZ_CALC_DFMAD_256(val1, val2);
		CZ_CALC_DFMAD_256(val1, val2);
		CZ_CALC_DFMAD_256(val1, val2);
		CZ_CALC_DFMAD_256(val1, val2);
	}

	arr[index] = val1 + val2;
}

__global__ void CZCudaCalcKernelInteger32(
	void *buf			/*!<[in] Data buffer. */
) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int *arr = (int*)buf;
	int val1 = index;
	int val2 = arr[index];
	int i;

	for(i = 0; i < CZ_CALC_BLOCK_LOOPS; i++) {
		CZ_CALC_IMAD32_256(val1, val2);
		CZ_CALC_IMAD32_256(val1, val2);
		CZ_CALC_IMAD32_256(val1, val2);
		CZ_CALC_IMAD32_256(val1, val2);
		CZ_CALC_IMAD32_256(val1, val2);
		CZ_CALC_IMAD32_256(val1, val2);
		CZ_CALC_IMAD32_256(val1, val2);
		CZ_CALC_IMAD32_256(val1, val2);
		CZ_CALC_IMAD32_256(val1, val2);
		CZ_CALC_IMAD32_256(val1, val2);
		CZ_CALC_IMAD32_256(val1, val2);
		CZ_CALC_IMAD32_256(val1, val2);
		CZ_CALC_IMAD32_256(val1, val2);
		CZ_CALC_IMAD32_256(val1, val2);
		CZ_CALC_IMAD32_256(val1, val2);
		CZ_CALC_IMAD32_256(val1, val2);
	}

	arr[index] = val1 + val2;
}

__global__ void CZCudaCalcKernelInteger24(
	void *buf			/*!<[in] Data buffer. */
) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int *arr = (int*)buf;
	int val1 = index;
	int val2 = arr[index];
	int i;

	for(i = 0; i < CZ_CALC_BLOCK_LOOPS; i++) {
		CZ_CALC_IMAD24_256(val1, val2);
		CZ_CALC_IMAD24_256(val1, val2);
		CZ_CALC_IMAD24_256(val1, val2);
		CZ_CALC_IMAD24_256(val1, val2);
		CZ_CALC_IMAD24_256(val1, val2);
		CZ_CALC_IMAD24_256(val1, val2);
		CZ_CALC_IMAD24_256(val1, val2);
		CZ_CALC_IMAD24_256(val1, val2);
		CZ_CALC_IMAD24_256(val1, val2);
		CZ_CALC_IMAD24_256(val1, val2);
		CZ_CALC_IMAD24_256(val1, val2);
		CZ_CALC_IMAD24_256(val1, val2);
		CZ_CALC_IMAD24_256(val1, val2);
		CZ_CALC_IMAD24_256(val1, val2);
		CZ_CALC_IMAD24_256(val1, val2);
		CZ_CALC_IMAD24_256(val1, val2);
	}

	arr[index] = val1 + val2;
}

__global__ void CZCudaCalcKernelInteger64(
	void *buf			/*!<[in] Data buffer. */
) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	long long *arr = (long long*)buf;
	long long val1 = index;
	long long val2 = arr[index];
	int i;

	for(i = 0; i < CZ_CALC_BLOCK_LOOPS; i++) {
		CZ_CALC_IMAD64_256(val1, val2);
		CZ_CALC_IMAD64_256(val1, val2);
		CZ_CALC_IMAD64_256(val1, val2);
		CZ_CALC_IMAD64_256(val1, val2);
		CZ_CALC_IMAD64_256(val1, val2);
		CZ_CALC_IMAD64_256(val1, val2);
		CZ_CALC_IMAD64_256(val1, val2);
		CZ_CALC_IMAD64_256(val1, val2);
		CZ_CALC_IMAD64_256(val1, val2);
		CZ_CALC_IMAD64_256(val1, val2);
		CZ_CALC_IMAD64_256(val1, val2);
		CZ_CALC_IMAD64_256(val1, val2);
		CZ_CALC_IMAD64_256(val1, val2);
		CZ_CALC_IMAD64_256(val1, val2);
		CZ_CALC_IMAD64_256(val1, val2);
		CZ_CALC_IMAD64_256(val1, val2);
	}

	arr[index] = val1 + val2;
}

static float CZCudaCalcDevicePerformanceTest(
	struct CZDeviceInfo *info,	/*!<[in,out] CUDA-device information. */
	int mode			/*!<[in] Run performance test in one of modes. */
) {
	CZDeviceInfoBandLocalData *lData;
	float timeMs = 0.0;
	float performanceKOPs = 0.0;
	hipEvent_t start;
	hipEvent_t stop;
	int blocksNum = info->heavyMode? info->core.muliProcCount: 1;
	int i;
	if(info == NULL)
		return 0;

	CZ_CUDA_CALL(hipEventCreate(&start),
		return 0);

	CZ_CUDA_CALL(hipEventCreate(&stop),
		hipEventDestroy(start);
		return 0);

	lData = (CZDeviceInfoBandLocalData*)info->band.localData;
	int threadsNum = info->core.maxThreadsPerBlock;
	if(threadsNum == 0) {
		int warpSize = info->core.SIMDWidth;
		if(warpSize == 0)
			warpSize = CZ_DEF_WARP_SIZE;
		threadsNum = warpSize * 2;
		if(threadsNum > CZ_DEF_THREADS_MAX)
			threadsNum = CZ_DEF_THREADS_MAX;
	}
	CZLog(CZLogLevelLow, "Starting %s test on %s on %d block(s) %d thread(s) each.",
		(mode == CZ_CALC_MODE_FLOAT)? "single-precision float":
		(mode == CZ_CALC_MODE_DOUBLE)? "double-precision float":
		(mode == CZ_CALC_MODE_INTEGER32)? "32-bit integer":
		(mode == CZ_CALC_MODE_INTEGER24)? "24-bit integer":
		(mode == CZ_CALC_MODE_INTEGER64)? "64-bit integer": "unknown",
		info->deviceName,
		blocksNum,
		threadsNum);
	for(i = 0; i < CZ_CALC_LOOPS_NUM; i++) {

		float loopMs = 0.0;
		CZ_CUDA_CALL(hipEventRecord(start, 0),
			hipEventDestroy(start);
			hipEventDestroy(stop);
			return 0);
		switch(mode) {
		case CZ_CALC_MODE_FLOAT:
			CZCudaCalcKernelFloat<<<blocksNum, threadsNum>>>(lData->memDevice1);
			break;

		case CZ_CALC_MODE_DOUBLE:
			CZCudaCalcKernelDouble<<<blocksNum, threadsNum>>>(lData->memDevice1);
			break;

		case CZ_CALC_MODE_INTEGER32:
			CZCudaCalcKernelInteger32<<<blocksNum, threadsNum>>>(lData->memDevice1);
			break;

		case CZ_CALC_MODE_INTEGER24:
			CZCudaCalcKernelInteger24<<<blocksNum, threadsNum>>>(lData->memDevice1);
			break;

		case CZ_CALC_MODE_INTEGER64:
			CZCudaCalcKernelInteger64<<<blocksNum, threadsNum>>>(lData->memDevice1);
			break;

		default: // WTF!
			hipEventDestroy(start);
			hipEventDestroy(stop);
			return 0;
		}

		CZ_CUDA_CALL(hipGetLastError(),
			hipEventDestroy(start);
			hipEventDestroy(stop);
			return 0);

		CZ_CUDA_CALL(hipEventRecord(stop, 0),
			hipEventDestroy(start);
			hipEventDestroy(stop);
			return 0);

		CZ_CUDA_CALL(hipEventSynchronize(stop),
			hipEventDestroy(start);
			hipEventDestroy(stop);
			return 0);

		CZ_CUDA_CALL(hipEventElapsedTime(&loopMs, start, stop),
			hipEventDestroy(start);
			hipEventDestroy(stop);
			return 0);

		timeMs += loopMs;
	}

	//CZLog(CZLogLevelLow, "Test complete in %f ms.", timeMs);
	performanceKOPs = (
		(float)info->core.muliProcCount *
		(float)CZ_CALC_LOOPS_NUM *
		(float)threadsNum *
		(float)CZ_CALC_BLOCK_LOOPS *
		(float)CZ_CALC_OPS_NUM *
		(float)CZ_CALC_BLOCK_SIZE *
		(float)CZ_CALC_BLOCK_NUM
	) / (float)timeMs;

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return performanceKOPs;
}

int CZCudaCalcDevicePerformance(
	struct CZDeviceInfo *info	/*!<[in,out] CUDA-device information. */
) {

	if(info == NULL)
		return -1;

	if(CZCudaCalcDevicePerformanceReset(info) != 0)
		return -1;

	if(!CZCudaIsInit())
		return -1;
	info->perf.calcFloat = CZCudaCalcDevicePerformanceTest(info, CZ_CALC_MODE_FLOAT);
	if(((info->major > 1)) ||
		((info->major == 1) && (info->minor >= 3)))
		info->perf.calcDouble = CZCudaCalcDevicePerformanceTest(info, CZ_CALC_MODE_DOUBLE);
	info->perf.calcInteger32 = CZCudaCalcDevicePerformanceTest(info, CZ_CALC_MODE_INTEGER32);
	info->perf.calcInteger24 = CZCudaCalcDevicePerformanceTest(info, CZ_CALC_MODE_INTEGER24);
	info->perf.calcInteger64 = CZCudaCalcDevicePerformanceTest(info, CZ_CALC_MODE_INTEGER64);

	return 0;
}

int main(int argc, char **argv){
	if(argc!=3 || std::string(argv[1])!="-d"){
		printf("./cuda -d num\t#num is the id of device\n");
		return 0;
	}
	int device_num, num;
	sscanf(argv[2], "%d", &num);
	device_num=CZCudaDeviceFound();
	if(num<0 || device_num <1 || num >= device_num){
		printf("device id error!\n");
		return 0;
	}
	for(int i=0; i<5; i++){
		CZDeviceInfo CZ_info;
		CZCudaIsInit();
		CZCudaReadDeviceInfo(&CZ_info, num);
		CZCudaCalcDeviceSelect(&CZ_info);
		CZCudaPrepareDevice(&CZ_info);
		CZCudaCalcDeviceBandwidthTest(&CZ_info);
		CZCudaCalcDevicePerformance(&CZ_info);
		CZCudaCleanDevice(&CZ_info);
		printf("device %d, %s, driver version:%s\n\n", num, CZ_info.deviceName, CZ_info.drvVersion);
		printf("H2D pin:%f GiB/s\n", CZ_info.band.copyHDPin/1024/1024);
		printf("D2H pin:%f GiB/s\n", CZ_info.band.copyDHPin/1024/1024);
		printf("D2D    :%f GiB/s\n", CZ_info.band.copyDD/1024/1024);
		printf("sp     :%f Gflop/s\n", CZ_info.perf.calcFloat/1000/1000);
		printf("dp     :%f Gflop/s\n", CZ_info.perf.calcDouble/1000/1000);
		printf("64-bit :%f Gflop/s\n", CZ_info.perf.calcInteger64/1000/1000);
		printf("32-bit :%f Gflop/s\n", CZ_info.perf.calcInteger32/1000/1000);
		printf("24-bit :%f Gflop/s\n\n", CZ_info.perf.calcInteger24/1000/1000);
	}
	return 0;
}

